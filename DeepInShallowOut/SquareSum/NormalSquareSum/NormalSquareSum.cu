#include "hip/hip_runtime.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include "../../../CudaHelper.h"

const unsigned int  DATE_SIZE = 1 << 24; // 16M

// Kernel function to compute square sum of an int array to a result 
__global__ void SquareSum(int *pInputData, int *pResult)
{
	int result = 0;
	int i = 0;
	for(i = 0; i < DATE_SIZE; i++)
	{
		result += pInputData[i] * pInputData[i];
	}

	*pResult = result;
}

int main(int argv, char* argc[])
{
	// Get cuda device count
	int iCount;
	hipGetDeviceCount(&iCount);
	if(0 == iCount)
	{
		printf("There is no cuda device\n");
		return false; 
	}

	// Find the first suitable device
	int i;
	for (i = 0; i < iCount; i++)
	{
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			// find a prop > CUDA 1.X device and break
			if(prop.major >= 1)
			{
				break;
			}
		}
	}

	// can not find a prop > CUDA 1.X device and return false
	if(i == iCount)
	{
		printf("There is no CUDA 1.X device\n");
		return false;
	}

	// Set the suitable device to current
	hipSetDevice(i);

	// Malloc host data
	int *pHostData = (int*)malloc(sizeof(int)*DATE_SIZE);
	int  hostResult = 0;
	if( 0 == pHostData)
	{
		printf("malloc host data failed!!!\n");
		return -1;
	}

	// Generate 16M rand data range from 0 to 4
	for(int i = 0; i < DATE_SIZE; i++)
	{
		pHostData[i] = rand() % 5;
	}

	// Malloc device data
	int *pDeviceData = NULL;
	int *pDeviceResult = NULL;
	
	HANDLE_CUDA_ERROR(hipMalloc((void**)&pDeviceData, sizeof(int) * DATE_SIZE));
	HANDLE_CUDA_ERROR(hipMalloc((void**)&pDeviceResult, sizeof(int)));

	printf("\nGPU COMPUTE BEGIN********************\n");

	// Record total time elapsed via GPU
	TIME_TRACE_CUDA_EVENT_START(TotalElpasedTimeViaGPU);

	// Copy host data to device
	TIME_TRACE_CUDA_EVENT_START(hipMemcpyHostToDevice);
	HANDLE_CUDA_ERROR(hipMemcpy(pDeviceData, pHostData, sizeof(int) * DATE_SIZE, hipMemcpyHostToDevice));
	TIME_TRACE_CUDA_EVENT_STOP(hipMemcpyHostToDevice);

	// Execute Kernel 
	TIME_TRACE_CUDA_EVENT_START(SqureSumKernel);
	SquareSum<<<1, 1>>>(pDeviceData, pDeviceResult);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(err));
	}
	TIME_TRACE_CUDA_EVENT_STOP(SqureSumKernel);

	// Copy result from device
	TIME_TRACE_CUDA_EVENT_START(hipMemcpyDeviceToHost);
	HANDLE_CUDA_ERROR(hipMemcpy(&hostResult, pDeviceResult, sizeof(int), hipMemcpyDeviceToHost)); 
	TIME_TRACE_CUDA_EVENT_STOP(hipMemcpyDeviceToHost);
	
	TIME_TRACE_CUDA_EVENT_STOP(TotalElpasedTimeViaGPU);

	// Free device memory
	HANDLE_CUDA_ERROR(hipFree(pDeviceData));
	HANDLE_CUDA_ERROR(hipFree(pDeviceResult));


	// Print result
	printf("Square Sum Computed Via Result GPU is %d.\n", hostResult);

	// hipDeviceReset to ensure Visual Profile run correctly
	HANDLE_CUDA_ERROR(hipDeviceReset());

	printf("\nGPU COMPUTE END********************\n");


	printf("\nCPU COMPUTE BEGIN********************\n");
	// Compute in CPU for comparision
	hostResult = 0;

	TIME_TRACE_CPU_START(TotalElpasedTimeViaCPU);
	for (int i = 0 ; i < DATE_SIZE; i++)
	{
		hostResult += pHostData[i] * pHostData[i];
	}
	TIME_TRACE_CPU_STOP(TotalElpasedTimeViaCPU);

	// Free host memory
	free(pHostData); pHostData = NULL;

	// Print result
	printf("Square Sum Computed Result Via CPU is %d.\n", hostResult);

	printf("\nCPU COMPUTE END********************\n");

	return 0;
	
}
