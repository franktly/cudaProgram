#include "hip/hip_runtime.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include "../../../CudaHelper.h"

const unsigned int  DATE_SIZE  = 1 << 24; // 16M 
const unsigned int  BLOCK_SIZE = 1024; // block size
/*
 * 
 * One Block: every thread in the block exec DATE_SIZE/BLOCK_SIZE computation task
 * 
// friendly for global memory access(data space locality and benefit for cache line), adjacent thread access adjacent data address space
 * thread k compute column k data:(k =  0 ~ BLOCK_SIZE)
 * 
 * ThreadId:     tid0                   tid1              ...   tidBLOCK_SIZE-1
 * ---------------------------------------------------------------------------------------------
 * DataId  :     dat0                   dat1              ...   datBLOCK_SIZE-1
 * DataId  :     datBLOCK_SIZE+0    datBLOCK_SIZE+1       ...   datBLOCK_SIZE+BLOCK_SIZE-1
 * DataId  :     datBLOCK_SIZE*2+0  datBLOCK_SIZE*2+1     ...   datBLOCK_SIZE*2+BLOCK_SIZE-1
 * 
 * ...
 * 
// badly for global memory access(data space locality and benefit for cache line), adjacent thread does not access adjacent data address space
 * thread k compute row k data:(k =  0 ~ BLOCK_SIZE)
 *																								ThreadId:
 * -------------------------------------------------------------------------------------------------------
 * DataId  :     dat0                   dat1              ...   datBLOCK_SIZE-1                  tid0
 * DataId  :     datBLOCK_SIZE+0    datBLOCK_SIZE+1       ...   datBLOCK_SIZE+BLOCK_SIZE-1       tid1
 * DataId  :     datBLOCK_SIZE*2+0  datBLOCK_SIZE*2+1     ...   datBLOCK_SIZE*2+BLOCK_SIZE-1     tid2
 * 
 * ... 
 */
// Kernel function to compute square sum of an int array to a result 
__global__ void SquareSum(int *pInputData, int *pResult)
{
	int result = 0;
	const int tid = threadIdx.x;
	int i = 0;
	// friendly for global memory access(data space locality and benefit for cache line), adjacent thread access adjacent data address space
	/*
	 * for(i = tid; i < DATE_SIZE; i = i + BLOCK_SIZE)
	 * {
	 *     result += pInputData[i] * pInputData[i];
	 * }
	 */

	// badly for global memory access, adjacent thread does not access adjacent data address space
	const int count = DATE_SIZE/BLOCK_SIZE;
	for(i = tid * count; i < (tid +1) * count; i++)
	{
		result += pInputData[i] * pInputData[i];
	}
	pResult[tid] = result;
}

int main(int argv, char* argc[])
{
	// Get cuda device count
	int iCount;
	hipGetDeviceCount(&iCount);
	if(0 == iCount)
	{
		printf("There is no cuda device\n");
		return false; 
	}

	// Find the first suitable device
	int i;
	for (i = 0; i < iCount; i++)
	{
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			// find a prop > CUDA 1.X device and break
			if(prop.major >= 1)
			{
				break;
			}
		}
	}

	// can not find a prop > CUDA 1.X device and return false
	if(i == iCount)
	{
		printf("There is no CUDA 1.X device\n");
		return false;
	}

	// Set the suitable device to current
	hipSetDevice(i);

	// Malloc host data
	int *pHostData = (int*)malloc(sizeof(int)*DATE_SIZE);
	int *pHostThreadData = (int*)malloc(sizeof(int)*BLOCK_SIZE);
	int  hostResult = 0;
	if( 0 == pHostData)
	{
		printf("malloc host data failed!!!\n");
		return -1;
	}

	// Generate 16M rand data range from 0 to 4
	for(int i = 0; i < DATE_SIZE; i++)
	{
		pHostData[i] = rand() % 5;
	}

	// Malloc device data
	int *pDeviceData = NULL;
	int *pDeviceResult = NULL;
	
	HANDLE_CUDA_ERROR(hipMalloc((void**)&pDeviceData, sizeof(int) * DATE_SIZE));
	HANDLE_CUDA_ERROR(hipMalloc((void**)&pDeviceResult, sizeof(int) * BLOCK_SIZE));

	printf("\nGPU COMPUTE BEGIN********************\n");

	// Record total time elapsed via GPU
	TIME_TRACE_CUDA_EVENT_START(TotalElpasedTimeViaGPU);

	// Copy host data to device
	TIME_TRACE_CUDA_EVENT_START(hipMemcpyHostToDevice);
	HANDLE_CUDA_ERROR(hipMemcpy(pDeviceData, pHostData, sizeof(int) * DATE_SIZE, hipMemcpyHostToDevice));
	TIME_TRACE_CUDA_EVENT_STOP(hipMemcpyHostToDevice);

	// Execute Kernel 
	TIME_TRACE_CUDA_EVENT_START(SqureSumKernel);
	SquareSum<<<1, BLOCK_SIZE>>>(pDeviceData, pDeviceResult);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(err));
	}
	TIME_TRACE_CUDA_EVENT_STOP(SqureSumKernel);

	// Copy result from device
	TIME_TRACE_CUDA_EVENT_START(hipMemcpyDeviceToHost);
	HANDLE_CUDA_ERROR(hipMemcpy(pHostThreadData, pDeviceResult, sizeof(int) * BLOCK_SIZE, hipMemcpyDeviceToHost)); 
	TIME_TRACE_CUDA_EVENT_STOP(hipMemcpyDeviceToHost);
	
	TIME_TRACE_CUDA_EVENT_STOP(TotalElpasedTimeViaGPU);

	// Free device memory
	HANDLE_CUDA_ERROR(hipFree(pDeviceData));
	HANDLE_CUDA_ERROR(hipFree(pDeviceResult));

	// Add every thread result in CPU
	TIME_TRACE_CPU_START(AddEveryThreadData);
	for (int i = 0 ; i < BLOCK_SIZE; i++)
	{
		hostResult += pHostThreadData[i];
	}
	TIME_TRACE_CPU_STOP(AddEveryThreadData);

	// Print result
	printf("Square Sum Computed Via Result GPU & CPU is %d.\n", hostResult);

	// hipDeviceReset to ensure Visual Profile run correctly
	HANDLE_CUDA_ERROR(hipDeviceReset());

	printf("\nGPU COMPUTE END********************\n");

	printf("\nCPU COMPUTE BEGIN********************\n");
	// Compute in CPU for comparision
	hostResult = 0;

	TIME_TRACE_CPU_START(TotalElpasedTimeViaCPU);
	for (int i = 0 ; i < DATE_SIZE; i++)
	{
		hostResult += pHostData[i] * pHostData[i];
	}
	TIME_TRACE_CPU_STOP(TotalElpasedTimeViaCPU);

	// Free host memory
	free(pHostThreadData); pHostThreadData = NULL;
	free(pHostData); pHostData = NULL;

	// Print result
	printf("Square Sum Computed Result Via CPU is %d.\n", hostResult);

	printf("\nCPU COMPUTE END********************\n");

	return 0;
	
}
